#include "hip/hip_runtime.h"
#include <defs.h>
#include <symbol.h>
#include <gpudefs.h>
#include <dvector.h>
#include <gpuconsts.cuh>

__global__ __launch_bounds__( WS * SOW, MBO )
void algSAT_stage4( float *g_out, const float *g_in) {

	const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y, col = bx*WS+tx, row0 = by*WS;

	__shared__ float s_block[ WS ][ WS+1 ];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx];

	g_in += (row0+ty)*c_width+col;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        **bdata = *g_in;
        bdata += SOW;
        g_in += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        **bdata = *g_in;
    }

	__syncthreads();

	if( ty == 0 ) {

        {   // calculate y -----------------------
            float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[0][tx];

            float prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                **bdata = prev = **bdata + prev;
        }

        {   // calculate x -----------------------
            float *bdata = s_block[tx];

            float prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                *bdata = prev = *bdata + prev;
        }

	}

	__syncthreads();

    bdata = (float (*)[WS+1]) &s_block[ty][tx];

	g_out += (row0+ty)*c_width+col;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        *g_out = **bdata;
        bdata += SOW;
        g_out += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        *g_out = **bdata;
    }

}

//-- Host ---------------------------------------------------------------------

__host__
void calc_borders( int& left,
                   int& top,
                   int& right,
                   int& bottom,
                   const int& w,
                   const int& h,
                   const int& extb ) {

    left = extb*WS;
    top = extb*WS;

    if( extb > 0 ) {

        right = (extb+1)*WS-(w%WS);
        bottom = (extb+1)*WS-(h%WS);

    } else {

        right = WS-(w%WS);
        if( right == WS ) right = 0;
        bottom = WS-(h%WS);
        if( bottom == WS ) bottom = 0;

    }

}

__host__
bool extend( const int& w,
             const int& h,
             const int& extb ) {
    return (w%32>0 or h%32>0 or extb>0);
}

__host__
void calc_alg_setup( alg_setup& algs,
                     const int& w,
                     const int& h ) {

    algs.width = w;
    algs.height = h;
    algs.m_size = (w+WS-1)/WS;
    algs.n_size = (h+WS-1)/WS;
    algs.last_m = algs.m_size-1;
    algs.last_n = algs.n_size-1;
    algs.border = 0;
    algs.carry_width = algs.m_size*WS;
    algs.carry_height = algs.n_size*WS;
    algs.carry_height = h;
    algs.inv_width = 1.f/(float)w;
    algs.inv_height = 1.f/(float)h;

}

__host__
void calc_alg_setup( alg_setup& algs,
                     const int& w,
                     const int& h,
                     const int& extb ) {

    int bleft, btop, bright, bbottom;
    calc_borders( bleft, btop, bright, bbottom, w, h, extb );

    algs.width = w;
    algs.height = h;
    algs.m_size = (w+bleft+bright+WS-1)/WS;
    algs.n_size = (h+btop+bbottom+WS-1)/WS;
    algs.last_m = (bleft+w-1)/WS;
    algs.last_n = (btop+h-1)/WS;
    algs.border = extb;
    algs.carry_width = algs.m_size*WS;
    algs.carry_height = algs.n_size*WS;
    algs.inv_width = 1.f/(float)w;
    algs.inv_height = 1.f/(float)h;

}


__host__
void prepare_algSAT( alg_setup& algs,
                     dvector<float>& d_inout,
                     const float *h_in,
                     const int& w,
                     const int& h )
{
    algs.width = w;
    algs.height = h;

    if( w % 32 > 0 ) algs.width += (32 - (w % 32));
    if( h % 32 > 0 ) algs.height += (32 - (h % 32));

    calc_alg_setup( algs, algs.width, algs.height );
    up_alg_setup( algs );

    d_inout.copy_from( h_in, w, h, algs.width, algs.height );
}

__host__
void algSAT( dvector<float>& d_out,
             const dvector<float>& d_in,
             const alg_setup& algs ) {

	const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
    const dim3 cg_img( algs.m_size, algs.n_size );
    const dim3 cg_ybar( nWm, 1 );
    const dim3 cg_vhat( 1, nHm );

    algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_out, d_in);
}
